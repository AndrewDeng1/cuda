
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

__global__ void addKernel(float* a, float* b, float* c, int N){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < N){
        c[idx]=a[idx]+b[idx];
    }
}

void launchAdd(float* a, float* b, float* c, int N){
    float *d_a, *d_b, *d_c;
    size_t size = N * sizeof(float);

    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    addKernel<<<(N+255)/256, 256>>>(d_a, d_b, d_c, N);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}