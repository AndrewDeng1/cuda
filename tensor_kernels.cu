
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

__global__ void addKernel(float* a, float* b, float* c, int N){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < N){
        c[idx]=a[idx]+b[idx];
    }
}

void launchAdd(float* a, float* b, float* c, int N){
    float *d_a, *d_b, *d_c;
    size_t size = N * sizeof(float);

    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    addKernel<<<(N+255)/256, 256>>>(d_a, d_b, d_c, N);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

__global__ void subtractKernel(float* a, float* b, float* c, int N){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < N){
        c[idx]=a[idx]-b[idx];
    }
}

void launchSubtract(float* a, float* b, float* c, int N){
    float *d_a, *d_b, *d_c;
    size_t size = N * sizeof(float);

    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    subtractKernel<<<(N+255)/256, 256>>>(d_a, d_b, d_c, N);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

__global__ void multiplyKernel(float* a, float* b, float* c, int N){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < N){
        c[idx]=a[idx]*b[idx];
    }
}

void launchMultiply(float* a, float* b, float* c, int N){
    float *d_a, *d_b, *d_c;
    size_t size = N * sizeof(float);

    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    multiplyKernel<<<(N+255)/256, 256>>>(d_a, d_b, d_c, N);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

__global__ void divideKernel(float* a, float* b, float* c, int N){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < N){
        c[idx]=a[idx]/b[idx];
    }
}

void launchDivide(float* a, float* b, float* c, int N){
    float *d_a, *d_b, *d_c;
    size_t size = N * sizeof(float);

    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    divideKernel<<<(N+255)/256, 256>>>(d_a, d_b, d_c, N);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}